#include "nccl.h"
#include <stdio.h>
#include "common.h"


int sendRecv(ncclUniqueId Id){
    ncclGroupStart();

    ncclGroupEnd();
}


int main(){
    // init communicator
    ncclUniqueId Id;
    ncclGetUniqueId(&Id);
    int nranks = 3;
    ncclGroupStart();
    for(int rank=0; rank<nranks; rank++){
        //hipSetDevice();
        //ncclCommInitRank(comm, nranks, Id,  rank);
    }
    ncclGroupEnd();

    printf("Running sendrecv.");
    sendRecv(Id);

    return 0;
}
